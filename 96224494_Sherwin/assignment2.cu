
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
using namespace std;

__global__  void AddArray(int* d_a,int* d_b, int* d_c,int col,int row)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int id=col*y+i;
    if(i < col &&y < row)
        d_c[id] = d_a[id] + d_b[id];
}
int main()
{
    int row,col;
    printf("enter row and col");
    scanf("%d%d",&row,&col);
     
 
    int h_a[row][col],h_b[row][col],h_c[row][col];
    int Array_Bytes = row*col* sizeof(int);  
    for(int i=0; i<row; i++)
    {
       for(int j=0;j<col;j++)
       {
            h_a[i][j] = i+j;
            h_b[i][j] = i+j;
        }
    }
    int *d_a,*d_b, *d_c;
    hipMalloc((void**)&d_b, Array_Bytes);
    hipMalloc((void**)&d_a, Array_Bytes);
    hipMalloc((void**)&d_c, Array_Bytes);
    // Copy the array from CPU (h_in) to the GPU (d_in)
    hipMemcpy(d_b, h_b, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a, h_a, Array_Bytes, hipMemcpyHostToDevice);
    AddArray<<<dim3(col,row,1),1 >>>(d_a,d_b,d_c,col,row);
    // Copy the resulting array from GPU (d_out) to the CPU (h_out)
    hipMemcpy(h_c, d_c, Array_Bytes, hipMemcpyDeviceToHost);
    for(int i=0; i<row; i++)
    {
       for(int j=0;j<col;j++)
       {
           printf("%d\t", h_c[i][j]);
        }
       printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}


