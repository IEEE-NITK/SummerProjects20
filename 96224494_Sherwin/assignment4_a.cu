
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
using namespace std;
__global__ void Array_max(int* d_out, int* d_array, int Size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ int sh_array[1024];
    // Shared memory that is exclusive for a block.
    // An array of size 1024 declared for common access to all the threads in a block
    // Each block has its own shared memory
   
    // Copy data from global to shared memory
    if(id < Size)
        sh_array[tid] = d_array[id];
    __syncthreads();
   
    // Perform parallel reduction in shared memory
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=Size || id+s>=Size)
            continue;
        if(tid<s)
            {
               if(sh_array[tid] < sh_array[tid + s])
                sh_array[tid]= sh_array[tid + s];
            }
        // Each iteration reduces size of active array by half
    }
    __syncthreads();
    // Only thread 0 of each block writes back the result of that block into global memory
    if(tid==0)
        d_out[bid] = sh_array[tid];  
}
__global__ void Array_min(int* d_out, int* d_array, int Size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ int sh_array[1024];
    // Shared memory that is exclusive for a block.
    // An array of size 1024 declared for common access to all the threads in a block
    // Each block has its own shared memory
   
    // Copy data from global to shared memory
    if(id < Size)
        sh_array[tid] = d_array[id];
    __syncthreads();
   
    // Perform parallel reduction in shared memory
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=Size || id+s>=Size)
            continue;
        if(tid<s)
            {
               if(sh_array[tid] > sh_array[tid + s])
                sh_array[tid]= sh_array[tid + s];
            }
        // Each iteration reduces size of active array by half
    }
    __syncthreads();
    // Only thread 0 of each block writes back the result of that block into global memory
    if(tid==0)
        d_out[bid] = sh_array[tid];  
}
int Find_max_GPU(int h_array[], int Size)
{
    int* d_array, *d_out, *d_sum;
    hipMalloc((void**)&d_array, Size*sizeof(int));
    hipMalloc((void**)&d_out, ceil(Size*1.0/1024)*sizeof(int));
    hipMalloc((void**)&d_sum, sizeof(int));
    hipMemcpy(d_array, h_array, sizeof(int) * Size, hipMemcpyHostToDevice);
    int h_sum;
    Array_max <<<ceil(Size*1.0/1024), 1024>>> (d_out, d_array, Size);
    Array_max <<<1, 1024>>> (d_sum, d_out, ceil(Size*1.0/1024));
    hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
    hipFree(d_out);
    hipFree(d_sum);
    return h_sum;
}

int Find_min_GPU(int h_array[], int Size)
{
    int* d_array, *d_out, *d_sum;
    hipMalloc((void**)&d_array, Size*sizeof(int));
    hipMalloc((void**)&d_out, ceil(Size*1.0/1024)*sizeof(int));
    hipMalloc((void**)&d_sum, sizeof(int));
    hipMemcpy(d_array, h_array, sizeof(int) * Size, hipMemcpyHostToDevice);
    int h_sum;
    Array_min <<<ceil(Size*1.0/1024), 1024>>> (d_out, d_array, Size);
    Array_min <<<1, 1024>>> (d_sum, d_out, ceil(Size*1.0/1024));
    hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
    hipFree(d_out);
    hipFree(d_sum);
    return h_sum;
}



int Find_min_CPU(int h_array[], int Size)
{
    int naive_min = h_array[0] ;
    for(int i=0; i<Size-1; i++)
         {
            if(h_array[i]>h_array[i+1])
            naive_min=h_array[i+1];
         }
    return naive_min;
}



int Find_max_CPU(int h_array[], int Size)
{
    int naive_max = h_array[0];
    for(int i=0; i<Size-1; i++)
         {
            if(h_array[i]<h_array[i+1])
            naive_max=h_array[i+1];
         }
    return naive_max;
}




int main()
{
    int Size;
    printf("Enter the array size\n");
    scanf("%d",&Size);
    int h_array[Size];
    for(int i=0; i<Size; i++)
        h_array[i] =i+1;
    int max = Find_max_GPU(h_array, Size);
    int min = Find_min_GPU(h_array, Size);
    int naive_min = Find_min_CPU(h_array, Size);
    int naive_max = Find_max_CPU(h_array, Size);
    printf("max no is %d\n",max);
    printf("min no is %d\n",min);
    if(max==naive_max&&min==naive_min)
        printf("Result computed correctly\n");
    else
        printf("Result wrong!");
   
   
}
