
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void mulArray(int* d_a,int* d_b, int* d_c,int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i <size)
        d_c[i] = d_a[i] * d_b[i];
}



__global__ void Array_Add(int* d_out, int* d_array, int Size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ int sh_array[1024];
    // Shared memory that is exclusive for a block.
    // An array of size 1024 declared for common access to all the threads in a block
    // Each block has its own shared memory
   
    // Copy data from global to shared memory
    if(id < Size)
        sh_array[tid] = d_array[id];
    __syncthreads();
   
    // Perform parallel reduction in shared memory
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=Size || id+s>=Size)
            continue;
        if(tid<s)
            sh_array[tid] += sh_array[tid + s];
        // Each iteration reduces size of active array by half
    }
    __syncthreads();
    // Only thread 0 of each block writes back the result of that block into global memory
    if(tid==0)
        d_out[bid] = sh_array[tid];  
}
int Find_Sum_GPU(int h_array[], int Size)
{
    int* d_array, *d_out, *d_sum;
    hipMalloc((void**)&d_array, Size*sizeof(int));
    hipMalloc((void**)&d_out, ceil(Size*1.0/1024)*sizeof(int));
    hipMalloc((void**)&d_sum, sizeof(int));
    hipMemcpy(d_array, h_array, sizeof(int) * Size, hipMemcpyHostToDevice);
    int h_sum;
    Array_Add <<<ceil(Size*1.0/1024), 1024>>> (d_out, d_array, Size);
    Array_Add <<<1, 1024>>> (d_sum, d_out, ceil(Size*1.0/1024));
    hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
    hipFree(d_out);
    hipFree(d_sum);
    return h_sum;
}






int main()
{
    int size;
    printf("enter array size");
    scanf("%d",&size);
     
 
    int h_a[size],h_b[size],h_c[size];
    int Array_Bytes = size* sizeof(int);  
    for(int i=0; i<size; i++)
    {
       
            h_a[i]= 2;
            h_b[i]= 1;
    }
     
     
    printf("hello\n");
    int *d_a,*d_b, *d_c;
    hipMalloc((void**)&d_b, Array_Bytes);
    hipMalloc((void**)&d_a, Array_Bytes);
    hipMalloc((void**)&d_c, Array_Bytes);
    // Copy the array from CPU (h_in) to the GPU (d_in)
    hipMemcpy(d_b, h_b, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a, h_a, Array_Bytes, hipMemcpyHostToDevice);
    mulArray<<<size,1 >>>(d_a,d_b,d_c,size);
    // Copy the resulting array from GPU (d_out) to the CPU (h_out)
    hipMemcpy(h_c, d_c, Array_Bytes, hipMemcpyDeviceToHost);
    int h_sum = Find_Sum_GPU(h_c, size);
    printf("dot product sum is %d",h_sum);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
