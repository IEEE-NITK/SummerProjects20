 #include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>  
using namespace std;
#define ll long long int
__global__ void Inclusive_Scan(ll *d_in, ll *d_out, ll Size, ll i)
{
    ll id = blockIdx.x * blockDim.x + threadIdx.x;
    ll step = 1 << i;
    if(id < Size)
    {
        if(id >= step)
        {
            if( d_in[id] < d_in[id-step])
                d_out[id] = d_in[id-step];
            else
                d_out[id] = d_in[id];
        }
        else
        {
            d_out[id] = d_in[id];
        }
    }
    __syncthreads();
}
int main()
{
    ll *h_in, *h_out;
    ll Size;
    printf("Enter size of the array.\n");
    scanf("%Ld",&Size);
    ll Array_Bytes = Size * sizeof(ll);
    h_in = (ll*)malloc(Array_Bytes);
    h_out = (ll*)malloc(Array_Bytes);
   
    // Populating input array with random numbers
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }
    printf("Input Array : \n");
    for(ll i=0; i<Size; i++)
        printf("%Ld\t",h_in[i]);
    printf("\n");
    ll *d_in, *d_out;
    hipMalloc((void**)&d_in, Array_Bytes);
    hipMalloc((void**)&d_out, Array_Bytes);
    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);
    ll iterations = (ll)floor(log2((double)Size)) + 1;
    for(ll i=0; i<iterations; i++)
    {
        Inclusive_Scan <<< (int)ceil(1.0*Size/1024), 1024>>> (d_in, d_out, Size, i);
        hipMemcpy(d_in, d_out, Array_Bytes, hipMemcpyDeviceToDevice);
    }
    hipMemcpy(h_out, d_out, Array_Bytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    printf("Inclusive Scan Array : \n");
    for(ll i=0; i<Size; i++)
        printf("%Ld\t",h_out[i]);
       
    printf("\n");
   
   
}
