#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#define ll long long int
const int Block_Size = 1024;
// This GPU kernel does blockwise in-place scan
__global__ void Blelloch_Exclusive_Scan(ll *d_in, ll* d_out)
{
    __shared__ ll sh_array[Block_Size];
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    // Copying data from global to shared memory
    sh_array[tid] = d_in[id];
    __syncthreads();
    /** Performing block-wise in-place Blelloch scan **/
    // First step of Blelloch scan : REDUCTION
    for(int k=2; k <= Block_Size; k *= 2)
    {
        if((tid+1) % k == 0)
        {
         
            if( sh_array[tid]  < sh_array[tid - (k/2)])
              sh_array[tid]= sh_array[tid - (k/2)];
       
        }
        __syncthreads();
    }
    // At the end of reduction, the last element of each block conatins the sum of all elements in that block
    // We store these block-wise sums in d_out
    if(tid == (Block_Size - 1))
    {
        d_out[bid] = sh_array[tid];
        sh_array[tid] = 0;
    }
    __syncthreads();
    // Second step of Blelloch scan : DOWNSWEEP
    // This is structurally the exact reverse of the reduction step
    for(int k = Block_Size; k >= 2; k /= 2)
    {
        if((tid+1) % k == 0)
        {
            ll temp = sh_array[tid - (k/2)];
            sh_array[tid - (k/2)] = sh_array[tid];
            if( sh_array[tid]  < temp)
              sh_array[tid]= temp;
        }
        __syncthreads();
    }
    // Copying the scan result back into global memory
    d_in[id] = sh_array[tid];
    // d_in now contains blockwise scan result
    __syncthreads();
}
// This GPU kernel adds the value d_out[id] to all values in the (id)th block of d_in
__global__ void Add(ll* d_in, ll* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    d_in[id] += d_out[bid];
    __syncthreads();
}
int main()
{
    ll *h_in, *h_scan;
    int Size;
    printf("Enter size of the array.\n");
    scanf("%Ld",&Size);
   
    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);
    int Array_Bytes = Size * sizeof(ll);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(ll);
    h_in = (ll*)malloc(Array_Bytes);
    h_scan = (ll*)malloc(Array_Bytes);
    // Populating array with random numbers
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
       h_in[i] = rand()%10;
       
    }
     printf("Input Array : \n");
    for(ll i=0; i<Size; i++)
        printf("%Ld\t",h_in[i]);
    printf("\n");
    ll *d_in, *d_out, *d_sum;
    // GPU Memory allocations
    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(ll));  
    // Padding the input array to the next multiple of Block_Size.  
    // The scan algorithm is not dependent on elements past the end of the array, so we don't have to use a special case for the last block.
    hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(ll));
    // Copying input array from CPU to GPU
    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);
    Blelloch_Exclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    // After first kernel call, d_in has the blockwise scan results and d_out is an auxiliary array that has the blockwise sums
    // Second kernel call is done to scan the blockwise sums array
    // Then the ith value in the resultant scanned blockwise sums array is added to every value in the ith block
    // This addition step is done in the Add() kernel
    // This is required only if size of the array is greater than the block size
    if(Size > Block_Size)
    {
        Blelloch_Exclusive_Scan <<< 1, Block_Size >>> (d_out, d_sum);
        Add <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }
    // Copying the result back to the CPU
    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    printf("Exclusive Scan Array : \n");
    for(ll i=0; i<Size; i++)
        printf("%Ld\t",h_scan[i]);
       
    printf("\n");
}
