
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

// GPU Code
// __global__ indicates that it is a GPU kernel, that can be called from the CPU
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
         int id = blockIdx.x * blockDim.x + threadIdx.x;
         if(id < N)
           
            d_c[id] = d_a[id] + d_b[id];
    
}

// CPU Code
int main()
{   
    int N;
    cout << "Enter the array size : ";
    cin >> N;
    float h_a[N], h_b[N], h_c[N];

    int Array_Bytes = N * sizeof(float);  

    for(int i=0; i<N; i++)
    {
        h_a[i] = i;
    }
    for(int i=0;i<N;i++)
    {
    h_b[i]=i+1;
    }
 

    // Declaring pointers for allocation on the device 
    float* d_a;
    float* d_b;
    float* d_c;

    // Allocating device memory
    hipMalloc((void**)&d_a,  Array_Bytes);
    hipMalloc((void**)&d_b,  Array_Bytes);
    hipMalloc((void**)&d_c,  Array_Bytes);

    // Copying input operands from host to device
    // For the GPU to perform any operation, the data has to be present in the GPU memory
    hipMemcpy(d_a, h_a,  Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b,  Array_Bytes, hipMemcpyHostToDevice);

    // Launching kernel with 1 block and 1 thread
    // The host launches the kernel on the device
   
    Add<<<ceil(1.0*N/1024), 1024>>>(d_a, d_b, d_c,N);

    // Copying the result from device to host
    hipMemcpy(h_c, d_c, Array_Bytes, hipMemcpyDeviceToHost);
     

   
    for(int i=0; i<N; i++)
        cout << h_c[i] << " ";

    hipFree(d_a);    
    hipFree(d_b);       
    hipFree(d_c);
}

