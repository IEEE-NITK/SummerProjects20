#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;






__global__ void MatrixAdd(float *a, float *b, float *c, int M,int N) {

   int col = threadIdx.x + blockIdx.x * blockDim.x;
   int row = threadIdx.y + blockIdx.y * blockDim.y;
   int index = col + row * N;
   if (col < N && row < M) {
       c[index] = a[index] + b[index];
   }
}



int main(){

  int *A, *B, *C;
  cout<<"Enter the number of row and column: ";
  cin>>M>>N;
  size_t dsize = M*N*sizeof(float);
  A = (float *)malloc(M*N*sizeof(float));
  B = (float *)malloc(M*N*sizeof(float));
  C = (float *)malloc(M*N*sizeof(float));

  for (int i = 0; i < M; i++)
    for (int j = 0; j < N; j++) {
      A[i][j] = 1.0;
      B[i][j] = 1.5;
      C[i][j] = 0.0;}

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, dsize);
  hipMalloc(&d_B, dsize);
  hipMalloc(&d_C, dsize);

  hipMemcpy(d_A, A, dsize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, dsize, hipMemcpyHostToDevice);

  dim3 dimBlock(16,16);
  dim3 dimGrid((int)ceil(M/dimBlock.x),(int)ceil(N/dimBlock.y));

  MatrixAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,M, N);
  hipMemcpy(C, d_C, dsize, hipMemcpyDeviceToHost);
  for (int i = 0; i < M; i++)
    for (int j = 0; j < N; j++)
    {
      cout<<C[i][j] <<" ";
     }
  
  return 0;
}
