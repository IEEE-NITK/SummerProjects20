
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;


__global__ void sum(float* d_a1, float* d_a2, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<size)
    d_a1[id]=d_a1[id]+d_a2[id];
}

int main()
{
    int array_size;
    cout<<"enter array size : ";
    cin>>array_size;
    float h_a1[array_size], h_a2[array_size];
    int array_bytes=array_size*sizeof(float);
    //cout<<"elements of first array : ";
    for(int i=0; i<array_size; i++)
    {
        h_a1[i]=i;
    }
    //cout<<"elements of second array : ";
    for(int i=0; i<array_size; i++)
    {
        h_a2[i]=i;
    }
    
    float *d_a1, *d_a2;
    hipMalloc((void**)&d_a1, array_bytes);
    hipMalloc((void**)&d_a2, array_bytes);
    
    hipMemcpy(d_a1, h_a1, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a2, h_a2, array_bytes, hipMemcpyHostToDevice);
    
    sum<<<ceil(1.0*array_size/1024), 1024>>>(d_a1, d_a2, array_size);
    hipMemcpy(h_a1, d_a1, array_bytes, hipMemcpyDeviceToHost);
    
    for(int i=0; i<array_size; i++)
        cout << h_a1[i] << " ";
    hipFree(d_a1);
    hipFree(d_a2);
}