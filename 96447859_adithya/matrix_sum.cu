
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__  void Sum(float* d_a,float* d_b, int r,int c)
{
    int x=blockIdx.x*blockDim.x + threadIdx.x;
    int y=blockIdx.y*blockDim.y + threadIdx.y;
    int index=c*y+x;
    if(x<c && y<r)
        d_a[index]=d_a[index]+d_b[index];
}

int main() 
{
    int r,c;
    cin>>r>>c;
    float h_a[r][c], h_b[r][c];
    int bytes=r*c*sizeof(float);
    float count=1.0;
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
	    h_a[i][j]=count;
	    h_b[i][j]=count;
	    count=count+1.0;
        }
    }
	
    float *d_a, *d_b;
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_a, bytes);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    Sum<<<dim3(c,r,1),1 >>>(d_a,d_b,r,c);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    
    for(int i=0; i<r; i++)
    {
        for(int j=0;j<c;j++)
        {
           cout<<h_a[i][j]<<" ";
        }
        cout<<"\n";
    }
    hipFree(d_a);
    hipFree(d_b);
}
