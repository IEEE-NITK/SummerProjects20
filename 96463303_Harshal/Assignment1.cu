
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Sum(float* d1_in, float* d2_in, float* d_out, int* d_arr_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < *d_arr_size)
    
    d_out[i] = d1_in[i] + d2_in[i];

}

int main()
{
	int arr_size;
    cout << "Enter array size : ";
    cin >> arr_size;
    float h1_in[arr_size], h_out[arr_size],h2_in[arr_size];
    int arr_bytes = arr_size * sizeof(float);  
    cout<<"Enter "<<arr_size<<" elements array 1 and array 2\n";
    for(int i=0; i<arr_size; i++)
    cin>>h1_in[i];
   
    for(int i=0; i<arr_size; i++)
    cin>>h2_in[i];

    float *d1_in, *d_out, *d2_in;
     int *d_arr_size;

    hipMalloc((void**)&d1_in, arr_bytes);
    hipMalloc((void**)&d2_in, arr_bytes);
    hipMalloc((void**)&d_out, arr_bytes);
    hipMalloc((void**)&d_arr_size,sizeof(float));

    hipMemcpy(d1_in, h1_in, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d2_in, h2_in, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_size, &arr_size, sizeof(float), hipMemcpyHostToDevice);

    Sum<<<ceil(1.0*arr_size/1024), 1024>>>(d1_in, d2_in, d_out,d_arr_size);

    hipMemcpy(h_out, d_out, arr_bytes, hipMemcpyDeviceToHost);
    cout<<"Sum of the 2 arrays is\n";
    for(int i=0; i<arr_size; i++)
        cout << h_out[i] << " ";
    hipFree(d1_in);
    hipFree(d2_in);
    hipFree(d_out);
    hipFree(d_arr_size);

    }
