
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int arr_size_x,arr_size_y;

__global__ void Sum(float* d_in1,float* d_in2, float* d_out,int* d_arr_size_x,int* d_arr_size_y)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
    int k = threadIdx.y + blockIdx.y * blockDim.y;

	int i = k + j * *d_arr_size_y;

    if (k < *d_arr_size_y && j < *d_arr_size_x) 
       d_out[i] = d_in1[i] + d_in2[i];
}
int main()
{
    cout << "Enter the array size (row , col) : ";
    cin >> arr_size_x >> arr_size_y;

    int arr_bytes = arr_size_x * sizeof(float) * arr_size_y;  

	float *h_in1, *h_in2, *h_out;

    h_in1 = (float*)malloc(arr_bytes);
    h_in2 = (float*)malloc(arr_bytes);
    h_out = (float*)malloc(arr_bytes);

    for(int i=0; i<arr_size_x; i++)
    {
		for(int j = 0; j < arr_size_y; j++)
			{ 
			h_in1[i*arr_size_y + j] = i + 0.1;
            h_in2[i*arr_size_y + j] = i + 0.2; 
			}
    }


    float *d_in1,*d_in2, *d_out;
	int *d_arr_size_x,*d_arr_size_y;

    hipMalloc((void**)&d_in1, arr_bytes);
	hipMalloc((void**)&d_in2, arr_bytes);
    hipMalloc((void**)&d_out, arr_bytes);
	hipMalloc((void**)&d_arr_size_x, sizeof(int));
	hipMalloc((void**)&d_arr_size_y, sizeof(int));

    hipMemcpy(d_in1, h_in1, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_size_y, &arr_size_y, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_arr_size_x, &arr_size_x, sizeof(int), hipMemcpyHostToDevice);

	 dim3 dimBlock(32, 32);
	 dim3 dimGrid((int)ceil(1.0*arr_size_x/dimBlock.x),(int)ceil(1.0*arr_size_y/dimBlock.y));

    Sum<<<dimGrid, dimBlock>>>(d_in1, d_in2, d_out,d_arr_size_x,d_arr_size_y);

    hipMemcpy(h_out, d_out, arr_bytes, hipMemcpyDeviceToHost);

	for(int i=0; i<arr_size_x; i++)
		{for(int j = 0; j < arr_size_y; j++)
			cout << h_out[i*arr_size_y + j]<< " ";
			cout << endl;
			}

    hipFree(d_in1);
	hipFree(d_in2);
    hipFree(d_out);
	hipFree(d_arr_size_x);
	hipFree(d_arr_size_y);
}

