
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int Array_Size;

__global__ void Min(float* d_in1, float* d_out,int* d_array_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	int t_id = threadIdx.x;
    int b_id = blockIdx.x;

	__shared__ float a[1024];

    if(id < *d_array_size)
     a[t_id] = d_in1[id];    

	__syncthreads();

	for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=*d_array_size || id+s>=*d_array_size)
            continue;
        if(t_id<s)
            {
               if(a[t_id] > a[t_id + s])
                a[t_id]= a[t_id + s];
            }
        // Each iteration reduces size of active array by half
    }
    __syncthreads();

	 if(t_id==0)
        d_out[b_id] = a[t_id];   
}
__global__ void Max(float* d_in1, float* d_out,int* d_array_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	int t_id = threadIdx.x;
    int b_id = blockIdx.x;

	__shared__ float a[1024];

    if(id < *d_array_size)
     a[t_id] = d_in1[id];    

	__syncthreads();

	for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=*d_array_size || id+s>=*d_array_size)
            continue;
        if(t_id<s)
            {
               if(a[t_id] < a[t_id + s])
                a[t_id] = a[t_id + s];
            }
        // Each iteration reduces size of active array by half
    }
    __syncthreads();

	 if(t_id==0)
        d_out[b_id] = a[t_id];   
}
float Find_min_CPU(float h_array[], int Size)
{
    float naive_min = h_array[0] ;
    for(int i=0; i<Size-1; i++)
         {
            if(h_array[i]>h_array[i+1])
            naive_min=h_array[i+1];
         }
    return naive_min;
}



float Find_max_CPU(float h_array[], int Size)
{
    float naive_max = h_array[0];
    for(int i=0; i<Size-1; i++)
         {
            if(h_array[i]<h_array[i+1])
            naive_max=h_array[i+1];
         }
    return naive_max;
}
int main()
{
    cout << "Enter the array size : ";
    cin >> Array_Size;

    float h_in1[Array_Size],h_min,h_max;
    int Array_Bytes = Array_Size * sizeof(float);  
	int Array_Bytes_1 = (int)ceil(1.0*Array_Bytes/1024);  

    for(int i=0; i<Array_Size; i++)
    {
        h_in1[i] = i + 1.5;
    }

    float *d_in1, *d_out, *d_min,*d_max;
	int *d_array_size,*d_array_size_1;

    hipMalloc((void**)&d_in1, Array_Bytes);
	hipMalloc((void**)&d_min, sizeof(float));
	hipMalloc((void**)&d_max, sizeof(float));
    hipMalloc((void**)&d_out, Array_Bytes);
	hipMalloc((void**)&d_array_size, sizeof(int));
	hipMalloc((void**)&d_array_size_1, sizeof(int));

    hipMemcpy(d_in1, h_in1, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_size, &Array_Size, sizeof(int), hipMemcpyHostToDevice);
	 hipMemcpy(d_array_size_1, &Array_Bytes_1, sizeof(int), hipMemcpyHostToDevice);
    Min<<<((int)ceil(1.0*Array_Size/1024)), 1024>>>(d_in1, d_out, d_array_size);
	Min<<<1, 1024>>>(d_out, d_min, d_array_size_1);

	Max<<<((int)ceil(1.0*Array_Size/1024)), 1024>>>(d_in1, d_out,d_array_size);
	Max<<<1, 1024>>>(d_out, d_max,d_array_size_1);

	float naive_min = Find_min_CPU(h_in1, Array_Size);
    float naive_max = Find_max_CPU(h_in1, Array_Size);

    hipMemcpy(&h_min, d_min, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);

	cout << " Max " << h_max << " Min " << h_min << endl;
	//cout << " Max " << naive_max << " Min " << naive_min << endl;
	if(h_max==naive_max&&h_min==naive_min)
        printf("Correct\n");
    else
        printf("Error\n");

    hipFree(d_in1);
	hipFree(d_min);
	hipFree(d_max);
    hipFree(d_out);
	hipFree(d_array_size);
}
