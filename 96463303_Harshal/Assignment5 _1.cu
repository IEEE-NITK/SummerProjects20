#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define ll long long int

const int Block_Size = 1024;

__global__ void Inclusive_Scan(ll *d_in, ll* d_out)
{
    __shared__ ll sh_array[Block_Size];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

   
    sh_array[tid] = d_in[id];

    __syncthreads();

    for(int step = 1; step <= Block_Size; step *= 2)
    {
        if(tid >= step)
        {
            ll temp = sh_array[tid-step];
            __syncthreads();
            sh_array[tid] =max( temp,sh_array[tid]);
        }
        __syncthreads();
    }
    __syncthreads();

    d_in[id] = sh_array[tid];
    __syncthreads();

     if(tid == (Block_Size - 1))
        d_out[bid] = d_in[id];

    __syncthreads();
}


__global__ void Add(ll* d_in, ll* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    if(bid > 0)
        d_in[id] = max(d_out[bid-1],d_in[id]);

    __syncthreads();
}

int main()
{
    ll *h_in, *h_scan;

    int Size;
    cout << "Enter size of array\n";
    cin >> Size;

    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);  
    int Array_Bytes = Size * sizeof(ll);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(ll);

    h_in = (ll*)malloc(Array_Bytes);
    h_scan = (ll*)malloc(Array_Bytes);

    //Random nos
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }

    ll *d_in, *d_out, *d_sum;

    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(ll));  
   
   hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(ll));

    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);

    Inclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
   
    if(Size > Block_Size)
    {
        Inclusive_Scan <<< 1, Block_Size>>> (d_out, d_sum);
        Compare <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }

    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

   
    ll *pref;
    pref = (ll*)malloc(Array_Bytes);
    pref[0] = h_in[0];
    for(ll i=1; i<Size; i++)
        pref[i] = max(pref[i-1] , h_in[i]);

    ll flag = 0;
    for(ll i=0; i<Size; i++)
    {
        if(h_scan[i] != pref[i])
        {
            flag = 1;
            break;
        }
    }
    if(flag == 0)
        cout << "Completed!\n";
}
