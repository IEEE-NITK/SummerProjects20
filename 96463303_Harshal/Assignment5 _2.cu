#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define ll long long int

const int Block_Size = 1024;


__global__ void Blelloch_Exclusive_Scan(ll *d_in, ll* d_out)
{
    __shared__ ll sh_array[Block_Size];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    
    sh_array[tid] = d_in[id];
    __syncthreads();


    //  REDUCTION
    for(int k=2; k <= Block_Size; k *= 2)
    {
        if((tid+1) % k == 0)
        {
            sh_array[tid] =max( sh_array[tid - (k/2)],sh_array[tid]);
        }
        __syncthreads();
    }

   
    if(tid == (Block_Size - 1))
    {
        d_out[bid] = sh_array[tid];
        sh_array[tid] = 0;
    }
    __syncthreads();

    // DOWNSWEEP 
    for(int k = Block_Size; k >= 2; k /= 2)
    {
        if((tid+1) % k == 0)
        {
            ll temp = sh_array[tid - (k/2)];
            sh_array[tid - (k/2)] = sh_array[tid];
            sh_array[tid] =max( temp,sh_array[tid]);
        }
        __syncthreads();
    }

   
    d_in[id] = sh_array[tid];

    __syncthreads();
}

__global__ void Add(ll* d_in, ll* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    d_in[id] = max(d_out[bid],d_in[id]);

    __syncthreads();
}

int main()
{
    ll *h_in, *h_scan;

    int Size;
    cout << "Enter size of the array.\n";
    cin >> Size;

    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);
    int Array_Bytes = Size * sizeof(ll);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(ll);
    h_in = (ll*)malloc(Array_Bytes);
    h_scan = (ll*)malloc(Array_Bytes);

    
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }

   

    ll *d_in, *d_out, *d_sum;

    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(ll));  
    hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(ll));

    
    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);

    Blelloch_Exclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
   
    
    if(Size > Block_Size)
    {
        Blelloch_Exclusive_Scan <<< 1, Block_Size >>> (d_out, d_sum);
        Add <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }

    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    ll *pref;
    pref = (ll*)malloc(Array_Bytes);
    pref[0] = 0;
    for(ll i=1; i<Size; i++)
        pref[i] = max(pref[i-1] , h_in[i-1]);

    
    ll flag = 0;
    for(ll i=0; i<Size; i++)
    {
        if(h_scan[i] != pref[i])
        {
            flag = 1;
            break;
        }
    }
    if(flag == 0)
        cout << "Complete!\n";
  
}
