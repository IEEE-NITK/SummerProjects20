
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Add(float *d_a,float *d_b,float *d_c,int r,int c){
    
    int i =blockIdx.x*blockDim.x+threadIdx.x;
    int j =blockIdx.y*blockDim.y+threadIdx.y;
    int k = i+j*c;
    //i is defined for horizontal traversal
    if(i<c && j<r){
        d_c[k]=d_a[k]+d_b[k];
    }    
}


int main()
{
    int r,c,i,j;
    cout<<"Enter the rows and columns\n";
    cin>>r>>c;
    float h_a[r][c],h_b[r][c],h_c[r][c];
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            h_a[i][j]=i+j+3;
            h_b[i][j]=i*j;
        }
    }
    float *d_a,*d_b,*d_c;
    hipMalloc((void**)&d_a, (r*c)*sizeof(float));
    hipMalloc((void**)&d_b, (r*c)*sizeof(float));
    hipMalloc((void**)&d_c, (r*c)*sizeof(float));

    hipMemcpy(d_a, h_a, r*c*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, r*c*sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil(1.0*c/dimBlock.x),(int)ceil(1.0*r/dimBlock.y));
    Add<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,r,c);
    hipMemcpy(h_c, d_c, (r*c)*sizeof(float), hipMemcpyDeviceToHost);

    cout<<"Sum of the 2 matrices is:\n";
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            printf("%.2f ",h_c[i][j]);
        }
        cout<<"\n";
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}