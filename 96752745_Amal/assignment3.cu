#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

__global__ void Transpose(int *d_a,int max){

int i = blockIdx.x*blockDim.x+threadIdx.x;
int j = blockIdx.y*blockDim.y+threadIdx.y;
int id1 = i+max*j;
int id2 = j+max*i;
__syncthreads();

if(i<max && j<max)
{
	int t = d_a[id1];
	__syncthreads();
	d_a[id1]=d_a[id2];
	__syncthreads();
	d_a[id2]=t;
}
	
}

int main()
{
	int r,c,i,j,max;
	cout<<"Enter the number of rows and columns:\n";
	cin>>r>>c;
	max=r>c?r:c;
	int h_a[max][max]={0};
	for(i=0;i<r;i++)
	{
		for(j=0;j<c;j++)
		h_a[i][j]=2*i+j;
	}
	int *d_a;
	hipMalloc((void**)&d_a, max*max*sizeof(int));

	hipMemcpy(d_a, h_a, max*max*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil(1.0*max/dimBlock.x), (int)ceil(1.0*max/dimBlock.y));
	Transpose<<<dimGrid,dimBlock>>>(d_a,max);
	hipMemcpy(h_a, d_a, max*max*sizeof(int), hipMemcpyDeviceToHost);
	cout<<"The transpose matrix is:\n";
	for(i=0;i<c;i++)
	{
		for(j=0;j<r;j++)
		cout<<h_a[i][j]<<" ";
		cout<<"\n";
	}

	hipFree(d_a);
	return 0;
}