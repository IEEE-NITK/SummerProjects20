
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Add(float *array1, float *array2, float *out)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	out[id] = array1[id] + array2[id];
	
}

int main()
{
	unsigned int array_Size;
		
	cout << "Enter the size of Array: ";
	cin >> array_Size;

	float h_array1[array_Size], h_array2[array_Size], h_out[array_Size];
	
	int array_Bytes = array_Size * sizeof(float);

	for(int i=0;i<array_Size;i++)
		h_array1[i] = i;

	for(int i=1,j=0;j<array_Size;j++){
		h_array2[j] = i;
		
	}

	float *d_array1, *d_array2, *d_out;
	
	hipMalloc((void**)&d_array1, array_Bytes);
	hipMalloc((void**)&d_array2,array_Bytes);
	hipMalloc((void**)&d_out,array_Bytes);
	
	hipMemcpy(d_array1, h_array1, array_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_array2, h_array2, array_Bytes, hipMemcpyHostToDevice);
	

	Add<<<ceil(1.0*array_Size/1024),1024>>>(d_array1, d_array2, d_out);

	hipError_t e = hipMemcpy(h_out, d_out, array_Bytes, hipMemcpyDeviceToHost);
	
	if(e!=hipSuccess)
        	cout <<"CUDA error copying to Host: " << hipGetErrorString(e) << endl;
	
	for(int i=0;i<array_Size;i++)
		cout<< i << ". " << h_out[i] << "\n";
	
	hipFree(d_array1);
	hipFree(d_array2);
	hipFree(d_out); 

}