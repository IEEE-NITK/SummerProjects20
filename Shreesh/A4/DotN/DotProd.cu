
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdlib.h>
using namespace std;

//Device code
__global__ void dotProduct(int* A, int* B, int* C, int size){
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	//term wise product 
	C[abs_id] = A[abs_id] * B[abs_id];
}

__global__ void dotProdSum(int* d_C, int* d_out, int size){
	int t_id = threadIdx.x; int b_id = blockIdx.x;
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ int sh_data[1024];
	if(abs_id < size){
		sh_data[t_id] = d_C[abs_id];
	}
	__syncthreads();
	for(unsigned int s = blockIdx.x/2; s > 0; s = s/2){
		if(abs_id >= size || abs_id + s >= size){
			continue;
		}
		__syncthreads();
		if(t_id < s){
			sh_data[t_id] += sh_data[t_id + s];
		}
		__syncthreads();
	}

	if(t_id == 0){
		d_out[b_id] = sh_data[t_id];
	}

}
//Host code
int cpuDot(int* h_A, int* h_B, int size);
int gpuDot(int* h_A, int* h_B, int size);
void populateRandom(int* h_in, int size, int seed);
void printArray(int* arr, int size);

int main(int argc, char const *argv[])
{
	int size = 10;
	// cout << "Enter N: "; cin >>  size;
	int naive, parallel;
	int h_A[size]; populateRandom(h_A, size, 0);
	int h_B[size]; populateRandom(h_B, size, 5);

	// cout << "h_A: ";
	// printArray(h_A, size);
	// cout << endl;
	// cout << "h_B: ";
	// printArray(h_B, size);

	naive = cpuDot(h_A, h_B, size);
	cout << "Naive dot: " << naive << endl;
	parallel = gpuDot(h_A, h_B, size);
	cout << "Parallel dot: " << parallel << endl;
	return 0;
}

int cpuDot(int* h_A, int* h_B, int size){
	int naive = 0;
	for (int i = 0; i < size; ++i)
	{
		naive += h_A[i]*h_B[i];
	}
	return naive;
}

int gpuDot(int* h_A, int* h_B, int size){
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;

	int d_out[size];
	int* d_sum = NULL;
	int parallel = 0;

	int array_bytes = size * sizeof(int);
	int reduced_size = (int)ceil(size*1.0/1024);

	hipMalloc((void**)&d_A, array_bytes); 
	hipMalloc((void**)&d_B, array_bytes);
	hipMalloc((void**)&d_C, array_bytes);
	hipMalloc((void**)&d_out, reduced_size);
	hipMalloc((void**)&d_sum, sizeof(int));

	hipMemcpy(d_A, h_A, array_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, array_bytes, hipMemcpyHostToDevice);

	int b = ceil(size * 1.0/1024);
	int t = 1024;

	//kernel call
	dotProduct<<<b, t>>>(d_A, d_B, d_C, size); //prod then sum
	dotProdSum<<<b, t, 1024*sizeof(int)>>>(d_C, d_out, size);
	dotProdSum<<<1, t, 1024*sizeof(int)>>>(d_out, d_sum, size);

	hipMemcpy(&parallel, &d_sum, array_bytes, hipMemcpyDeviceToHost);
	return parallel;
}

void populateRandom(int* h_in, int size, int seed){
	srand(seed);
	for (int i = 0; i < size; ++i)
	{
		int random = rand() % 10;
		h_in[i] = random;
	}
}

void printArray(int* arr,int size){
	for (int i = 0; i < size; ++i)
	{
		cout << arr[i] << ", ";
	}
}

