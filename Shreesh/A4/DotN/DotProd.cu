
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdlib.h>
using namespace std;

//Device code
__global__ void dotProduct(int* A, int* B, int* C, int size){
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	//term wise product in global memory 
	C[abs_id] = A[abs_id] * B[abs_id];
}

__global__ void dotProdSum(int* d_C, int* d_out, int size){
	
	int t_id = threadIdx.x; int b_id = blockIdx.x;
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ int sh_data[1024];
	__syncthreads();
	
	//global -> shared memory 
	if(abs_id < size){
		sh_data[t_id] = d_C[abs_id];
	}
	__syncthreads();

	//reduce operation
	for(unsigned int s = blockDim.x/2; s > 0; s = s/2){
		__syncthreads();
		if(abs_id >= size || abs_id + s >= size){
			continue;
		}
		__syncthreads();

		if(t_id < s){
			sh_data[t_id] += sh_data[t_id + s];
		}
		__syncthreads();
	}

	if(t_id == 0){
		//each shared memory (per block) -> global array
		d_out[b_id] = sh_data[t_id];
	}

}
//Host code
int cpuDot(int* h_A, int* h_B, int size);
int gpuDot(int* h_A, int* h_B, int size);
void populateRandom(int* h_in, int size, int seed);
void printArray(int* arr, int size);

int main(int argc, char const *argv[])
{
	int size;
	cout << "Enter N: "; cin >>  size;
	int naive, parallel; bool ans = 0;
	int h_A[size]; populateRandom(h_A, size, 0); 
	int h_B[size]; populateRandom(h_B, size, 5);

	if(size > 10){
		cout << "Size of arrays too large." << endl;
		cout << "Do you still want me to display? (1/0):";
		cin >> ans;
	}
	if(ans==1){
		cout << "Array A: ";
		printArray(h_A, size);
		cout << endl;
		cout << "Array B: ";
		printArray(h_B, size);
	}
	naive = cpuDot(h_A, h_B, size);
	cout << "\n\nNaive dot: " << naive << endl;
	parallel = gpuDot(h_A, h_B, size);
	cout << "Parallel dot: " << parallel << endl;
	return 0;
}

int cpuDot(int* h_A, int* h_B, int size){
	int naive = 0;
	for (int i = 0; i < size; ++i)
	{
		naive += h_A[i]*h_B[i];
	}
	return naive;
}

int gpuDot(int* h_A, int* h_B, int size){
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* d_out = NULL;
	int* d_sum = NULL;
	int parallel = 0;

	int array_bytes = size * sizeof(int);
	int reduced_size = (int)ceil(size*1.0/1024);
	int reduced_bytes = reduced_size * sizeof(int);

	hipMalloc((void**)&d_A, array_bytes); 
	hipMalloc((void**)&d_B, array_bytes);
	hipMalloc((void**)&d_C, array_bytes);
	hipMalloc((void**)&d_out, reduced_bytes);
	hipMalloc((void**)&d_sum, sizeof(int));

	hipMemcpy(d_A, h_A, array_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, array_bytes, hipMemcpyHostToDevice);

	int b = ceil(size * 1.0/1024);
	int t = 1024;

	// int h_C[size];
	// cudaMemcpy(h_C, d_C, array_bytes, cudaMemcpyDeviceToHost);
	// for (int i = 0; i < size; ++i)
	// {
	// 	cout << "h_C: " << h_C[i] << endl;
	// }

	//kernel call - product then sum
	dotProduct<<<b, t>>>(d_A, d_B, d_C, size); 

	dotProdSum<<<b, t>>>(d_C, d_out, size);
	dotProdSum<<<1, t>>>(d_out, d_sum, reduced_size);

	hipMemcpy(&parallel, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	// parallel = 1;
	return parallel;
}

void populateRandom(int* h_in, int size, int seed){
	srand(seed);
	for (int i = 0; i < size; ++i)
	{
		int random = rand() % 10;
		h_in[i] = random;
	}
}

void printArray(int* arr,int size){
	for (int i = 0; i < size; ++i)
	{
		cout << arr[i] << ", ";
	}
}

