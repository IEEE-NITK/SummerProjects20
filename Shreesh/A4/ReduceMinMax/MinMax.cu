#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <stdlib.h>
using namespace std;

int size = 1024*1024; //2^20 elements

//Device code
__global__ void findMin(int* d_out, int* d_in, int size){
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	int t_id = threadIdx.x; 
	int b_id = blockIdx.x;

	__shared__ int sdata[1024];
	__syncthreads();

	//Copying data; global --> shared 
	if(abs_id < size){
		sdata[t_id] = d_in[abs_id];
		//there is one sdata array for every block
	}
	__syncthreads();

	//parallel reduce in shared memory
	for(unsigned int s = 1024/2; s > 0; s = s/2){
		//make sure all local s are initialized 
		__syncthreads();
		if(abs_id >= size || abs_id+s >= size)
			continue; 
		//make sure all unmapped threads are skipped 
		__syncthreads();

		if(t_id < s){
			if(sdata[t_id] > sdata[t_id + s]){
				//if +s is smaller then replace 
				sdata[t_id] = sdata[t_id + s];
			}
		}

		__syncthreads(); //All half comparisions are completed 
	} //each iteration reduces size of active array by half

	//Make sure all sdata[] have been reduced to size 1
	__syncthreads();

	if(t_id==0){
		d_out[b_id] = sdata[t_id];
	}
}

__global__ void findMax(int* d_out, int* d_in, int size){
	int abs_id = threadIdx.x + blockDim.x * blockIdx.x;
	int t_id = threadIdx.x; 
	int b_id = blockIdx.x;

	__shared__ int sdata[1024];
	__syncthreads();

	//Copying data; global --> shared 
	if(abs_id < size){
		sdata[t_id] = d_in[abs_id];
		//there is one sdata array for every block
	}
	__syncthreads();

	//parallel reduce in shared memory
	for(unsigned int s = 1024/2; s > 0; s = s/2){
		//make sure all local s are initialized 
		__syncthreads();
		if(abs_id >= size || abs_id+s >= size)
			continue; 
		//make sure all unmapped threads are skipped 
		__syncthreads();

		if(t_id < s){
			if(sdata[t_id] < sdata[t_id + s]){
				//if +s is greater then replace 
				sdata[t_id] = sdata[t_id + s];
			}
		}

		__syncthreads(); //All half comparisions are completed 
	} //each iteration reduces size of active array by half

	//Make sure all sdata[] have been reduced to size 1
	__syncthreads();

	if(t_id==0){
		d_out[b_id] = sdata[t_id];
	}
}

//Host code
void populateRandom(int* arr);
void printArray(int* arr);
void cpuMinMax(int* arr);
void gpuMinMax(int* h_in);
void compareResult(int* gpu, int* cpu);

//Driver function
int main(int argc, char const *argv[])
{
	int s = size;
	int h_in[s]; 
	populateRandom(h_in);
	cpuMinMax(h_in);
	gpuMinMax(h_in);
	return 0;
}

void gpuMinMax(int* h_in){

	int array_bytes = size * sizeof(int);
	int reduced_size = (int)ceil(size*1.0/1024);
	int reduced_bytes = reduced_size * sizeof(int);
	int* d_in  = NULL; //input array
	int* d_out = NULL; //reduced array
	int* d_min = NULL; //min
	int* d_max = NULL; //max
	int min, max;

	hipMalloc((void**)&d_in, array_bytes);
	hipMalloc((void**)&d_out,reduced_bytes);
	hipMalloc((void**)&d_min, sizeof(int));
	hipMalloc((void**)&d_max, sizeof(int));

	hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);
	int b = ceil(size*1.0/1024);
	//find min
	findMin<<<b, 1024, 1024*sizeof(int)>>>(d_out, d_in, size);
	findMin<<<1, 1024, 1024*sizeof(int)>>>(d_min, d_out, ceil(size*1.0/1024));
	hipMemcpy(&min, d_min, sizeof(int), hipMemcpyDeviceToHost);
	//find max
	findMax<<<b, 1024, 1024*sizeof(int)>>>(d_out, d_in, size);
	findMax<<<1, 1024, 1024*sizeof(int)>>>(d_max, d_out, ceil(size*1.0/1024));
	hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);
	//result
	cout << "\nReducing using GPU" << endl;
	cout << "Min: " << min << " | Max: " << max << endl;

	//free gpu memory 
	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_max);
	hipFree(d_min);

}

void populateRandom(int* h_in){
	unsigned int t = time(NULL);
	srand(t);
	for (int i = 0; i < size; ++i)
	{
		int random = rand();
		h_in[i] = random;
	}
}

void printArray(int* arr){
	for (int i = 0; i < size; ++i)
	{
		cout << arr[i] << ", ";
	}
}

void cpuMinMax(int* arr){
	int min, max;
	min = arr[0];
	max = arr[0];
	for (int i = 0; i < size; ++i)
	{
		if(min > arr[i])
			min = arr[i];
		if(max < arr[i])
			max = arr[i];
	}

	cout << "\nReducing using CPU" << endl;	
	cout << "Min: " << min << " | Max: " << max << endl;

}

