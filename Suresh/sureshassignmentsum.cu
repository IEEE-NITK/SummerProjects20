//assignment of arrays sum


#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void Sum(float* d1_in, float* d2_in, float* d_out, int* d_arraysize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < *d_arraysize)
        d_out[id] = d1_in[id] + d2_in[id];
}
int main()
{
    int h_arraysize;
    h_arraysize=100000;
    float h1_in[h_arraysize], h_out[h_arraysize] , h2_in[h_arraysize];
    int Array_Bytes = h_arraysize * sizeof(int);  
    for(int i=0; i<h_arraysize; i++)
    {
        h1_in[i]=i;
    }
    for(int i=0; i<h_arraysize;i++)
    {
	 h2_in[i]=i;
    }
    float *d1_in,*d2_in,*d_out;
    int *d_arraysize;
    hipMalloc((void**)&d1_in, Array_Bytes);
    hipMalloc((void**)&d2_in, Array_Bytes);
    hipMalloc((void**)&d_out, Array_Bytes);
    hipMalloc((void**)&d_arraysize,sizeof(int));
    // Copy the array from CPU (h_in) to the GPU (d_in)
    hipMemcpy(d1_in, h1_in, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d2_in, h2_in, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_arraysize, &h_arraysize, sizeof(int), hipMemcpyHostToDevice);
    Sum <<<ceil(1.0*h_arraysize/1024), 1024>>>(d1_in,d2_in,d_out,d_arraysize);
    // Copy the resulting array from GPU (d_out) to the CPU (h_out)
    hipMemcpy(h_out, d_out, Array_Bytes, hipMemcpyDeviceToHost);
    for(int i=h_arraysize-5; i<h_arraysize; i++)
        cout << h_out[i] << " ";
    hipFree(d1_in);
    hipFree(d2_in);
    hipFree(d_out);
    hipFree(d_arraysize);
}


