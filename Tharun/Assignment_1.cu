
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int Array_Size;

__global__ void Sum(float* d_in1,float* d_in2, float* d_out,int* d_array_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < *d_array_size)
     d_out[id] = d_in1[id] + d_in2[id];    
}
int main()
{
    cout << "Enter the array size : ";
    cin >> Array_Size;
	
    float h_in1[Array_Size],h_in2[Array_Size],h_out[Array_Size];
    int Array_Bytes = Array_Size * sizeof(float);  
	
    for(int i=0; i<Array_Size; i++)
    {
        h_in1[i] = i + 0.1;
		h_in2[i] = i + 0.2;
    }
	
    float *d_in1,*d_in2, *d_out;
	int *d_array_size;
	
    hipMalloc((void**)&d_in1, Array_Bytes);
	hipMalloc((void**)&d_in2, Array_Bytes);
    hipMalloc((void**)&d_out, Array_Bytes);
	hipMalloc((void**)&d_array_size, sizeof(int));

    hipMemcpy(d_in1, h_in1, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_size, &Array_Size, sizeof(int), hipMemcpyHostToDevice);
	
    Sum<<<ceil(1.0*Array_Size/1024), 1024>>>(d_in1, d_in2, d_out,d_array_size);
	
    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in1);
	hipFree(d_in2);
    hipFree(d_out);
	hipFree(d_array_size);
}
