
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void mul(float* d_a,float* d_b, float* d_c,int *size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id <*size)
        d_c[id] = d_a[id] * d_b[id];
}

__global__ void Array_Add(float* d_out, float* d_array, float Size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ float sh_array[1024];
    // Shared memory that is exclusive for a block. 
    // An array of size 1024 declared for common access to all the threads in a block
    // Each block has its own shared memory
    
    // Copy data from global to shared memory
    if(id < Size)
        sh_array[tid] = d_array[id];
    __syncthreads();
    
    // Perform parallel reduction in shared memory
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=Size || id+s>=Size)
            continue;
        if(tid<s)
            sh_array[tid] += sh_array[tid + s];
        // Each iteration reduces size of active array by half
    }
    __syncthreads();
    // Only thread 0 of each block writes back the result of that block into global memory
    if(tid==0)
        d_out[bid] = sh_array[tid];   
}
float Find_Sum_GPU(float h_array[], int Size)
{
    float* d_array, *d_out, *d_sum;
    hipMalloc((void**)&d_array, Size*sizeof(float));
    hipMalloc((void**)&d_out, ceil(Size*1.0/1024)*sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));
    hipMemcpy(d_array, h_array, sizeof(float) * Size, hipMemcpyHostToDevice);
    float h_sum;
    Array_Add <<<ceil(Size*1.0/1024), 1024>>> (d_out, d_array, Size);
    Array_Add <<<1, 1024>>> (d_sum, d_out, ceil(Size*1.0/1024));
    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_array);
    hipFree(d_out);
    hipFree(d_sum);
    return h_sum;
}
float Find_Sum_CPU(float h_array[], int Size)
{
    float naive_sum = 0;
    for(int i=0; i<Size; i++)
        naive_sum = naive_sum + h_array[i]; 
    return naive_sum;
}
int main()
{
    int Size;
    cout << "\nEnter the size of the array : ";
    cin >> Size;
    
	
	int Array_Bytes = Size* sizeof(float);  
	float h_in1[Array_Bytes],h_in2[Array_Bytes],h_out[Array_Bytes];
    for(int i=0; i<Size; i++)
       {
		h_in1[i] = i + 1.5;
		h_in2[i] = i + 2.5;
		}
		
		float *d_in1,*d_in2, *d_out;
	int *d_array_size;
	
    hipMalloc((void**)&d_in1, Array_Bytes);
	hipMalloc((void**)&d_in2, Array_Bytes);
    hipMalloc((void**)&d_out, Array_Bytes);
	hipMalloc((void**)&d_array_size, sizeof(int));
	
	
    hipMemcpy(d_in1, h_in1, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_size, &Array_Bytes, sizeof(int), hipMemcpyHostToDevice);

	mul<<<ceil(1.0*Size/1024), 1024 >>>(d_in1,d_in2,d_out,d_array_size);
	
	hipMemcpy(h_out, d_out, Array_Bytes, hipMemcpyDeviceToHost);
	
    float h_sum = Find_Sum_GPU(h_out, Size);
    float naive_sum = Find_Sum_CPU(h_out, Size);
	
    cout << "\nThe sum is " << h_sum << endl;
    if(h_sum == naive_sum)
        cout << "Result computed correctly.";
    else
        cout << "Result wrong!";
}
